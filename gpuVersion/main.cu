#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <stdio.h>

#define WARP_SIZE 32
#define WORD_MAX_SIZE 32
#define NUMBER_OF_BANKS WARP_SIZE
#define WARP_WORDS_SIZE (WARP_SIZE*WORD_MAX_SIZE)

__device__ int countBits(int a) {
    int result = 0;
    while(a) {
        result += a & 1;
        a>>= 1;
    }
    return result;
}

__global__ void compute(int* d_mem, int n, int l, int* d_pairs) {
    extern __shared__ int shm[];

    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    int wid = threadIdx.x / WARP_SIZE;
    int idInWarp = threadIdx.x % WARP_SIZE;

    if(gid >= n)
        return;

    for(int i = 0; i < WORD_MAX_SIZE; i++) {
        shm[i*NUMBER_OF_BANKS + idInWarp + wid*WARP_WORDS_SIZE] = d_mem[i + gid*WORD_MAX_SIZE];
    }

    int numberOfPairs = 0;

    for(int i = gid + 1; i < n; i++) {
        int distance = 0;
        for(int j = 0; j < WORD_MAX_SIZE; j++) {
            int temp = d_mem[j + i*WORD_MAX_SIZE] ^ shm[j*NUMBER_OF_BANKS + idInWarp + wid*WARP_WORDS_SIZE];
            int cd = countBits(temp);
            distance += cd;
        }
        if(distance == 1) {
            d_pairs[numberOfPairs + l*gid] = i;
            numberOfPairs++;
        }
    }
}

int main() {
    int n, l;
    std::cin >> n >> l;

    const int NUMBER_OF_THREADS = 1024;
    const int NUMBER_OF_BLOCKS = n / 1024 + 1;

    int* mem = new int[WORD_MAX_SIZE * n];
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < WORD_MAX_SIZE; j++) {
            mem[j + i*WORD_MAX_SIZE] = 0;
        }
    }
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < l; j++) {
            char a;
            do {
                std::cin.get(a);
            } while(isspace(a));
            mem[j / 32 + i * WORD_MAX_SIZE] *= 2;
            if(a == '1')
                mem[j / 32 + i * WORD_MAX_SIZE]++;
        }
    }
    int* d_mem;
    int* d_pairs;
    hipMalloc(&d_mem, WORD_MAX_SIZE*n*sizeof(int));
    hipMalloc(&d_pairs, l*n*sizeof(int));
    hipMemcpy(d_mem, mem, WORD_MAX_SIZE*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_pairs, 0, l*n*sizeof(int));

    size_t shmSize = 32 * 1024;
    compute<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS, shmSize>>>(d_mem, n, l, d_pairs);

    int* pairs = new int[n * l];

    hipMemcpy(pairs, d_pairs, n*l*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++) {
        int j = 0;
        while(pairs[j + i*l] != 0 && j < l) {
            std::cout << i << " " << pairs[j + i*l] << "\n";
            j++;
        }
    }


    delete[] pairs;
    delete[] mem;
    hipFree(d_mem);
    hipFree(d_pairs);
    return 0;
}

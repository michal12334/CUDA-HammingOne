#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <stdio.h>

struct Pair {
    int x;
    int y;
};

__device__ Pair hammingDistance(char* a, char* b, int l) {
    Pair result;
    result.x = 0;
    for(int i = 0; i < l; i++) {
        if(a[i] != b[i]) {
            result.x++;
            result.y = i;
        }
    }
    return result;
}

__global__ void compute(char* d_mem, int n, int l, bool* d_isAnswer) {
    int tid = threadIdx.x + blockIdx.x;
    for(int i = tid + 1; i < n; i++) {
        auto hd = hammingDistance(d_mem + tid*l, d_mem + i*l, l);
        if(hd.x == 1) {
            d_isAnswer[tid * l + hd.y] = true;
        }
    }
}

int main() {
    int n, l;
    std::cin >> n >> l;
    const int NUMBER_OF_THREADS = 1024;
    const int NUMBER_OF_BLOCKS = n / 1024 + 1;
    char* mem = new char[n * l + 1];
    bool* isAnswer = new bool[n * l];
    for(int i = 0; i < n; i++)
        std::cin >> (mem + i * l);
    char* d_mem;
    bool* d_isAnswer;
    hipMalloc(&d_mem, n * l * sizeof(char));
    hipMalloc(&d_isAnswer, n * l * sizeof(bool));
    hipMemcpy(d_mem, mem, n * l * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_isAnswer, 0, n * l * sizeof(bool));

    compute<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS>>>(d_mem, n, l, d_isAnswer);

    hipMemcpy(isAnswer, d_isAnswer, n * l * sizeof(bool), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++) {
        for(int j = 0; j < l; j++) {
            if(isAnswer[i * l + j]) {
                for(int k = 0; k < l; k++) {
                    std::cout << mem[i*l + k];
                }
                std::cout << " ";
                for(int k = 0; k < l; k++) {
                    if(k == j) {
                        std::cout << (mem[i*l + k] == '1' ? '0' : '1');
                        continue;
                    }
                    std::cout << mem[i*l + k];
                }
                std::cout << "\n";
            }
        }
    }

    delete[] mem;
    return 0;
}

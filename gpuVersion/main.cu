#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <stdio.h>

struct Pair {
    int x;
    int y;
};

__device__ Pair hammingDistance(char* a, char* b, int l) {
    Pair result;
    result.x = 0;
    for(int i = 0; i < l; i++) {
        if(a[i] != b[i]) {
            result.x++;
            result.y = i;
        }
    }
    return result;
}

__global__ void compute(char* d_mem, int n, int l, int* d_pairs, int* d_numberOfPairs) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = tid + 1; i < n; i++) {
        auto hd = hammingDistance(d_mem + tid*l, d_mem + i*l, l);
        if(hd.x == 1) {
            d_pairs[d_numberOfPairs[tid] + tid*l] = i;
            d_numberOfPairs[tid]++;
        }
    }
}

int main() {
    int n, l;
    std::cin >> n >> l;
    const int NUMBER_OF_THREADS = 1024;
    const int NUMBER_OF_BLOCKS = n / 1024 + 1;
    char* mem = new char[n * l + 1];
    int* pairs = new int[n * l];
    int* numberOfPairs = new int[n];
    for(int i = 0; i < n; i++)
        std::cin >> (mem + i * l);
    char* d_mem;
    int* d_pairs;
    int* d_numberOfPairs;
    hipMalloc(&d_mem, n * l * sizeof(char));
    hipMalloc(&d_pairs, n * l * sizeof(int));
    hipMalloc(&d_numberOfPairs, n * sizeof(int));
    hipMemcpy(d_mem, mem, n * l * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_numberOfPairs, 0, n * sizeof(int));

    compute<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS>>>(d_mem, n, l, d_pairs, d_numberOfPairs);

    hipMemcpy(pairs, d_pairs, n * l * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(numberOfPairs, d_numberOfPairs, n* sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++) {
        for(int j = 0; j < numberOfPairs[i]; j++) {
            for(int k = 0; k < l; k++)
                std::cout << mem[k + i*l];
            std::cout << " ";
            for(int k = 0; k < l; k++)
                std::cout << mem[k + pairs[j + i*l]*l];
            std::cout << "\n";
        }
    }

    delete[] mem;
    return 0;
}

#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <stdio.h>

struct Pair {
    int x;
    int y;
};

__device__ Pair hammingDistance(char* a, char* b, int l) {
    Pair result;
    result.x = 0;
    for(int i = 0; i < l; i++) {
        if(a[i] != b[i]) {
            result.x++;
            result.y = i;
        }
    }
    return result;
}

__global__ void compute(char* d_mem, int n, int l, bool* d_pairs) {
    extern __shared__ char shm[];
    int mem1Offset = 32*l;

    int tidX = threadIdx.x;
    int tidY = threadIdx.y;
    int tid = tidX + tidY * 32;
    int bidX = blockIdx.x;
    int bidY = blockIdx.y;

    if(tidX >= n || tidY >= n)
        return;
    
    int a = bidX * 32 * n * l;
    int b = bidY * 32 * n * l;
    for(int i = tid; i < mem1Offset; i+= blockDim.x*blockDim.y) {
        int index = i;
        shm[index] = d_mem[a + index/32 + (i%32)*l];
        shm[index + mem1Offset] = d_mem[b + index/32 + (i%32)*l];
    }

    __syncthreads();

    if(tidX >= tidY)
        return;

    Pair hm;
    hm.x = 0;
    for(int i = 0; i < l; i++) {
        if(shm[tidX + 32*i] != shm[tidY + 32*i + mem1Offset]) {
            hm.x++;
            hm.y = i;
        }
    }
    if(hm.x == 1) {
        d_pairs[a + tidX*l + hm.y] = true;
    }
}

int main() {
    int n, l;
    std::cin >> n >> l;
    const int NUMBER_OF_THREADS = 1024;
    const int NUMBER_OF_BLOCKS = n / 1024 + 1;
    char* mem = new char[n * l + 1];
    bool* pairs = new bool[n * l];
    for(int i = 0; i < n; i++)
        std::cin >> (mem + i * l);
    char* d_mem;
    bool* d_pairs;
    hipMalloc(&d_mem, n * l * sizeof(char));
    hipMalloc(&d_pairs, n * l * sizeof(bool));
    hipMemcpy(d_mem, mem, n * l * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_pairs, 0, n * l * sizeof(bool));

    size_t shmSize = (64*l) * sizeof(char);
    // size_t shmSize = 1e9;
    compute<<<dim3(sqrt(n)/32 + 1, sqrt(n)/32 + 1, 1), dim3(32, 32, 1), shmSize>>>(d_mem, n, l, d_pairs);

    hipMemcpy(pairs, d_pairs, n * l * sizeof(bool), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++) {
        for(int j = 0; j < l; j++) {
            if(pairs[j + i*l]) {
                for(int k = 0; k < l; k++)
                    std::cout << mem[k + i*l];
                std::cout << " ";
                for(int k = 0; k < l; k++)
                    if(k == j)
                        std::cout << (mem[k + i*l] == '0' ? '1' : '0');
                    else
                        std::cout << mem[k + i*l];
                std::cout << "\n";
            }
        }
    }

    delete[] mem;
    return 0;
}

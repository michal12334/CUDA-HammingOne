
#include <hip/hip_runtime.h>
#include <iostream>

#define WARP_SIZE 32
#define WORD_MAX_SIZE 32
#define NUMBER_OF_BANKS WARP_SIZE
#define WARP_WORDS_SIZE (WARP_SIZE*WORD_MAX_SIZE)
#define NUMBER_OF_THREADS 256
#define MAX_WORDS_IN_SHM 256

__global__ void compute(int* d_mem, int n, int l, int* d_pairs) {
    extern __shared__ int shm[];

    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    int wid = threadIdx.x / WARP_SIZE;
    int idInWarp = threadIdx.x % WARP_SIZE;
    int minGid = blockIdx.x * NUMBER_OF_THREADS;
    int tid = threadIdx.x;

    if(gid >= n)
        return;

    int numberOfPairs = 0;

    int word[WORD_MAX_SIZE];

    for(int i = 0; i < WORD_MAX_SIZE; i++) {
        word[i] = d_mem[i + gid*WORD_MAX_SIZE];
    }

    for(int i = minGid + 1; i < n; i += MAX_WORDS_IN_SHM) {
        int s = min(MAX_WORDS_IN_SHM, n - i);
        __syncthreads();
        if(tid < s) {
            for(int j = 0; j < WORD_MAX_SIZE; j++) {
                shm[j*NUMBER_OF_BANKS + idInWarp + wid*WARP_WORDS_SIZE] = d_mem[j + (tid + i) * WORD_MAX_SIZE];
            }
        }
        for(int j = 0; j < s; j++) {
            if(gid < j + i) {
                int distance = 0;
                for(int k = 0; k < WORD_MAX_SIZE; k++) {
                    int temp = word[k] ^ shm[k*NUMBER_OF_BANKS + (j%WARP_SIZE) + (j/WARP_SIZE)*WARP_WORDS_SIZE];
                    int cd = __popc(temp);
                    distance += cd;
                }
                if(distance == 1) {
                    d_pairs[numberOfPairs + l*gid] = i + j;
                    numberOfPairs++;
                }
            }
        }
    }
}

int main() {
    int n, l;
    std::cin >> n >> l;

    const int NUMBER_OF_BLOCKS = n / NUMBER_OF_THREADS + 1;

    int* mem = new int[WORD_MAX_SIZE * n];
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < WORD_MAX_SIZE; j++) {
            mem[j + i*WORD_MAX_SIZE] = 0;
        }
    }
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < l; j++) {
            char a;
            do {
                std::cin.get(a);
            } while(isspace(a));
            mem[j / (8*sizeof(int)) + i * WORD_MAX_SIZE] *= 2;
            if(a == '1')
                mem[j / (8*sizeof(int)) + i * WORD_MAX_SIZE]++;
        }
    }
    int* d_mem;
    int* d_pairs;
    hipMalloc(&d_mem, WORD_MAX_SIZE * n * sizeof(int));
    hipMalloc(&d_pairs, l * n * sizeof(int));
    hipMemcpy(d_mem, mem, WORD_MAX_SIZE * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_pairs, 0, l * n * sizeof(int));

    size_t shmSize = WORD_MAX_SIZE * MAX_WORDS_IN_SHM * sizeof(int);
    compute<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS, shmSize>>>(d_mem, n, l, d_pairs);

    int* pairs = new int[n * l];

    hipMemcpy(pairs, d_pairs, n * l * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++) {
        int j = 0;
        while(pairs[j + i*l] != 0 && j < l) {
            std::cout << i << " " << pairs[j + i*l] << "\n";
            j++;
        }
    }


    delete[] pairs;
    delete[] mem;
    hipFree(d_mem);
    hipFree(d_pairs);
    return 0;
}
